// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    cuda_hello<<<1, 1 >>> ();
    return 0;
}